
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdlib.h>
#include<time.h>


#define N 999999
#define nblocks 100
using namespace std;

__global__ void cudaArrayMax(float *a, float *b)
{
    int id = threadIdx.x + blockDim.x *blockIdx.x;
    int stride = nblocks;
    __shared__ float cache[nblocks];

    float thmax = a[id];
    for (int i = id; i < N; i += stride)
        if (a[i] > thmax)
            thmax = a[i];

    cache[threadIdx.x] = thmax;
    __syncthreads();

    float max = cache[0];
    for (int i = 0; i < nblocks; i++)
        if (cache[i] > max)
            max = cache[i];

    b[blockIdx.x] = max;

}

int main()
{
    srand(time(0));
    float *ha, *ht, hmax;
    float *da, *dt, dmax;
    unsigned int size = N *sizeof(float);

    ha = (float *) malloc(size);
    ht = (float *) malloc(nblocks *sizeof(float));

    for (int i = 0; i < N; i++)
        ha[i] = rand() / (float) RAND_MAX;

    /* ----- BEGIN CPU ----- */
    
    clock_t cpustart = clock();
    
    hmax = ha[0];
    for (int i = 0; i < N; i++)
        if (ha[i] > hmax)
            hmax = ha[i];
    
    clock_t cpuend = clock();    
    float cputime = 1000 *(cpuend - cpustart) / (float) CLOCKS_PER_SEC;

    cout << "cpu max is " << hmax << " in " << cputime << endl;

    /* ----- END CPU ----- */    

    /* ----- BEGIN GPU ----- */
    
    hipMalloc((void **) &da, size);
    hipMalloc((void **) &dt, nblocks *sizeof(float));

    clock_t gpustart = clock();
    
    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);

    cudaArrayMax <<<nblocks, nblocks>>> (da, dt);

    hipMemcpy(ht, dt, nblocks *sizeof(float), hipMemcpyDeviceToHost);

    dmax = ht[0];
    for (int i = 0; i < nblocks; i++)
        if (ht[i] > dmax)
            dmax = ht[i];
    
    clock_t gpuend = clock();
    float gputime = 1000 *(gpuend - gpustart) / (float) CLOCKS_PER_SEC;

    cout << "gpu max is " << dmax << " in " << gputime << endl;

    /* ----- END GPU ----- */

    cout << "speedup = " << cputime / gputime;

    hipFree(da);
    hipFree(dt);

    return 0;

} 
